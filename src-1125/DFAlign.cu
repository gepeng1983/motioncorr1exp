#include "hip/hip_runtime.h"
#include "DFAlign.h"
#include "func.h"
#include "mfunc.h"
#include "safefft.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


CDFAlign::CDFAlign(void)
{
	m_nsam=0;
	m_nsamRaw=0;
	m_iter=0;
	m_iterthres=0.0;
	//For image output
	m_bufIm=new float[(DISPDIM+2)*DISPDIM];
	m_dispIm=new short[DISPDIM*DISPDIM];

	m_bufFFTCorr=0;
	m_dispFFTCorr=new short[DISPDIM*DISPDIM];

	m_bufFFTRaw=0;
	m_dispFFTRaw=new short[DISPDIM*DISPDIM];

	m_bufCCMap=0;;

	initFFTWLock();

}


CDFAlign::~CDFAlign(void)
{
	if(m_bufIm!=0) delete [] m_bufIm;
	if(m_dispIm!=0) delete [] m_dispIm;

	if(m_bufFFTCorr!=0) delete [] m_bufFFTCorr;
	if(m_dispFFTCorr!=0) delete [] m_dispFFTCorr;

	if(m_bufFFTRaw!=0) delete [] m_bufFFTRaw;
	if(m_dispFFTRaw!=0) delete [] m_dispFFTRaw;

	if(m_bufCCMap!=0) delete [] m_bufCCMap;

	freeFFTWLock();
}


void CDFAlign::Message(const char *str)
{
	printf("%s\n",str);
}

void CDFAlign::UpdateDisplay()
{}

void CDFAlign::TextOutput(const char *str)
{
	m_log=str;
	printf("%s",m_log.c_str());
	//SendMessage(m_dlgwnd, WM_TSHOWLOG, 0,0);
	
	if(m_para.bSaveLog)
	{
		FILE *fp=fopen(m_fnLog,"a");
		fprintf(fp,"%s",m_log.c_str());
		fclose(fp);
	}
}

void* CDFAlign::ImageOutputThread(void *p)
{
	CDFAlign *pThis=(CDFAlign *)p;

	ifft2d(pThis->m_bufIm,DISPDIM);

	int size=DISPDIM*DISPDIM;
	float *buf=new float[size];
	fft2buf(buf,pThis->m_bufIm,DISPDIM);
	buf2DispShort(pThis->m_dispIm, buf, size);

	//add output code here
	//SendMessage(pThis->m_dlgwnd, WM_TSHOWIMAGE, 0,0);
	MRC mrc;
	mrc.open(pThis->m_dispCorrSum,"wb");
	mrc.createMRC(pThis->m_dispIm,DISPDIM,DISPDIM,1);
	mrc.close();
	
	delete [] buf;
	
	return (void *)0;
}
void CDFAlign::ImageOutput(float *buf)
{
	memcpy(m_bufIm,buf,sizeof(float)*(DISPDIM+2)*DISPDIM);
	
	pthread_t tid;
	int terror;
	terror=pthread_create(&tid,NULL,ImageOutputThread,(void *)this);
  	if(terror!=0)
  	{
		TextOutput("Error: Failed to create pthread: Image Output\n");;
   	return;
   }
   m_tids.push_back(tid);

}

void* CDFAlign::FFTOutputCorrThread(void *p)
{
	CDFAlign *pThis=(CDFAlign *)p;
	int nsam=pThis->m_nsam;
	float *buf=new float[(nsam+2)*nsam];
	FFTModulusToDispBuf(pThis->m_bufFFTCorr, buf, nsam);
	BinFFTDispBufToChar(pThis->m_dispFFTCorr, DISPDIM, buf, nsam);
	delete [] buf;

	//add output code here
	//SendMessage(pThis->m_dlgwnd, WM_TSHOWFFTCORR, 0,0);
	MRC mrc;
	mrc.open(pThis->m_dispCorrFFT,"wb");
	mrc.createMRC(pThis->m_dispFFTCorr,DISPDIM,DISPDIM,1);
	mrc.close();
	
	return (void *)0;
}
void CDFAlign::FFTOutputCorr(float *buf)
{
	if(m_bufFFTCorr==0) m_bufFFTCorr=new float[(m_nsam/2+1)*m_nsam];
	memcpy(m_bufFFTCorr,buf,sizeof(float)*(m_nsam/2+1)*m_nsam);
	
	pthread_t tid;
	int terror;
	terror=pthread_create(&tid,NULL,FFTOutputCorrThread,(void *)this);
  	if(terror!=0)
  	{
		TextOutput("Error: Failed to create pthread: FFT Output Corr\n");;
   	return;
   }
   m_tids.push_back(tid);
}

void* CDFAlign::FFTOutputRawThread(void *p)
{
	CDFAlign *pThis=(CDFAlign *)p;
	int nsam=pThis->m_nsam;
	float *buf=new float[(nsam+2)*nsam];
	FFTModulusToDispBuf(pThis->m_bufFFTRaw, buf, nsam);
	BinFFTDispBufToChar(pThis->m_dispFFTRaw, DISPDIM, buf, nsam);
	
	//add output code here
	//SendMessage(pThis->m_dlgwnd, WM_TSHOWFFTRAW, 0,0);
	MRC mrc;
	mrc.open(pThis->m_dispRawFFT,"wb");
	mrc.createMRC(pThis->m_dispFFTRaw,DISPDIM,DISPDIM,1);
	mrc.close();

	delete [] buf;
	
	return (void *)0;
}
void CDFAlign::FFTOutputRaw(float *buf)
{
	if(m_bufFFTRaw==0) m_bufFFTRaw=new float[(m_nsam/2+1)*m_nsam];
	memcpy(m_bufFFTRaw,buf,sizeof(float)*(m_nsam/2+1)*m_nsam);
	
	//pthread_t tid;
	//int terror;
	//terror=pthread_create(&tid,NULL,FFTOutputRawThread,(void *)this);
  	//if(terror!=0)
  	//{
	//	TextOutput("Error: Failed to create pthread: FFT Output Raw\n");;
   	//return;
        //}
        //m_tids.push_back(tid);
        FFTOutputRawThread( (void *) this);
}

void* CDFAlign::CCMapOutputThread(void *p)
{
	CDFAlign *pThis=(CDFAlign *)p;
	
	//add output code here
	//SendMessage(pThis->m_dlgwnd, WM_TSHOWCCMAP, 0,0);
	
	return (void *)0;
}
void CDFAlign::CCMapOutput(float *buf, void *pki)
{
	Vector<double> &ki=*(Vector<double> *)pki;
	if(m_bufCCMap!=0) delete [] m_bufCCMap;
	int size=m_para.CCPeakSearchDim*m_para.CCPeakSearchDim*ki.size();
	m_bufCCMap=new float[size];
	memcpy(m_bufCCMap,buf,sizeof(float)*size);
	m_kiCCMap.clear();
	for(int i=0;i<ki.size();i++) m_kiCCMap.push_back(ki[i]);

	pthread_t tid;
	int terror;
	terror=pthread_create(&tid,NULL,CCMapOutputThread,(void *)this);
  	if(terror!=0)
  	{
		TextOutput("Error: Failed to create pthread: CC Map Output\n");;
   	return;
   }
   m_tids.push_back(tid);

}

void CDFAlign::PlotFSC(float2* hRaw0, float2 *hRaw1, float2 *hCorr0, float2 *hCorr1,
					MASK *pPosList, int nsam, complex<double> direction)
{
	const int step=nsam/400;
	

	int nsamc=nsam/2+1;
	int sizec=nsamc*nsam;
	int nbox=nsamc/step+1;

	int i,id;
	float r,angle;
	float edge=cos(PI/4);

	float3 *fRaw0=new float3[nbox];   // x:cos(phase) y:amp^2 z:amp^2, along drift
	float3 *fRaw1=new float3[nbox];   // x:cos(phase) y:amp^2 z:amp^2, perpendicular to drift
	float3 *fCorr0=new float3[nbox];   // x:cos(phase) y:amp^2 z:amp^2, along drift
	float3 *fCorr1=new float3[nbox];   // x:cos(phase) y:amp^2 z:amp^2, perpendicular to drift
	memset(fRaw0,0,sizeof(float3)*nbox);
	memset(fRaw1,0,sizeof(float3)*nbox);
	memset(fCorr0,0,sizeof(float3)*nbox);
	memset(fCorr1,0,sizeof(float3)*nbox);

	if(abs(direction)>0.1) direction/=abs(direction);
	else direction=1.0;

	hipComplex a,b;

	
	for(i=0;i<sizec;i++)
	{
		r=sqrt(float(pPosList[i].x*pPosList[i].x+pPosList[i].y*pPosList[i].y));
		if(int(r)<=0 || r>=nsamc) continue;
		id=int(r/step);

		angle=fabs(pPosList[i].x*direction.real()+pPosList[i].y*direction.imag())/r;
		
		if(angle>edge) //along drift
		{
			a=hRaw0[i];
			b=hRaw1[i];
			fRaw0[id].x+=a.x*b.x+a.y*b.y;
			fRaw0[id].y+=a.x*a.x+a.y*a.y;
			fRaw0[id].z+=b.x*b.x+b.y*b.y;

			a=hCorr0[i];
			b=hCorr1[i];
			fCorr0[id].x+=a.x*b.x+a.y*b.y;
			fCorr0[id].y+=a.x*a.x+a.y*a.y;
			fCorr0[id].z+=b.x*b.x+b.y*b.y;
		}
		else
		{
			a=hRaw0[i];
			b=hRaw1[i];
			fRaw1[id].x+=a.x*b.x+a.y*b.y;
			fRaw1[id].y+=a.x*a.x+a.y*a.y;
			fRaw1[id].z+=b.x*b.x+b.y*b.y;

			a=hCorr0[i];
			b=hCorr1[i];
			fCorr1[id].x+=a.x*b.x+a.y*b.y;
			fCorr1[id].y+=a.x*a.x+a.y*a.y;
			fCorr1[id].z+=b.x*b.x+b.y*b.y;
		}
	}

	m_fscRaw0.resize(nbox,0.0);
	m_fscRaw1.resize(nbox,0.0);
	m_fscCorr0.resize(nbox,0.0);
	m_fscCorr1.resize(nbox,0.0);


	float t;
	for(i=0;i<nbox;i++)
	{
		t=fRaw0[i].y*fRaw0[i].z;
		if(t>0.00001) m_fscRaw0[i]=complex<double>(double(i)/nbox,fRaw0[i].x/sqrt(t));
		else m_fscRaw0[i]=complex<double>(double(i)/nbox,0.0);

		t=fRaw1[i].y*fRaw1[i].z;
		if(t>0.00001) m_fscRaw1[i]=complex<double>(double(i)/nbox,fRaw1[i].x/sqrt(t));
		else m_fscRaw1[i]=complex<double>(double(i)/nbox,0.0);

		t=fCorr0[i].y*fCorr0[i].z;
		if(t>0.00001) m_fscCorr0[i]=complex<double>(double(i)/nbox,fCorr0[i].x/sqrt(t));
		else m_fscCorr0[i]=complex<double>(double(i)/nbox,0.0);

		t=fCorr1[i].y*fCorr1[i].z;
		if(t>0.00001) m_fscCorr1[i]=complex<double>(double(i)/nbox,fCorr1[i].x/sqrt(t));
		else m_fscCorr1[i]=complex<double>(double(i)/nbox,0.0);
	}

	//add output code here
	//SendMessage(m_dlgwnd, WM_TSHOWFSC, 0,0);

	//output
	
	char str[512];
	TextOutput("\nFSC parallel(D) and perpendicular(U) to drift direction:\n");
	TextOutput("   Nq%       D_Raw       U_Raw       D_Corr       U_Corr\n");
	for(i=0;i<nbox;i++)
	{
		sprintf(str,"%7.2f %12.4f %12.4f %12.4f %12.4f\n",i*100.0/nbox,
			m_fscRaw0[i].imag(),m_fscRaw1[i].imag(),m_fscCorr0[i].imag(),m_fscCorr1[i].imag());
		TextOutput(str);
	}
	TextOutput("\n");


	delete [] fRaw0;
	delete [] fRaw1;
	delete [] fCorr0;
	delete [] fCorr1;
}

void CDFAlign::Done()
{
	   
	//add output code here
	//SendMessage(m_dlgwnd, WM_TDONE, 0,0);
	
}

void CDFAlign::PlotOutput(vector<complex<double> > &xy)
{
	m_curve=xy;
	//add output code here
	//SendMessage(m_dlgwnd, WM_TSHOWCURVE, 0,0);

}

int CDFAlign::getNFrame()
{
	MRC mrc;
	MRCZ mrcz;
	MRCT mrct;
	if(isMRC(m_fnStack) && mrc.open(m_fnStack,"rb") > 0) 
	{
		int n=mrc.getNz();
		mrc.close();
		return n;
	}
	else if (isMRCZ(m_fnStack) && mrcz.open(m_fnStack,"rb") > 0) 
        {
                int n=mrcz.getNz();
                mrcz.close();
                return n;
        }
	else if (isTIFF(m_fnStack) && mrct.open(m_fnStack,"rb") > 0) 
        {
                int n=mrct.getNz();
                mrct.close();
                return n;
        }
	return 0;
}
int CDFAlign::getNFrame(const char *filename)
{
	MRC mrc;
	MRCZ mrcz;
	MRCT mrct;
	if(isMRC(filename) && mrc.open(filename,"rb") > 0) 
	{
		int n=mrc.getNz();
		mrc.close();
		return n;
	}
	else if (isMRCZ(filename) && mrcz.open(filename,"rb") > 0)
        {
                int n=mrcz.getNz();
                mrcz.close();
                return n;
        }
	else if (isTIFF(filename) && mrct.open(filename,"rb") > 0)
        {
                int n=mrct.getNz();
                mrct.close();
                return n;
        }
	return 0;
}

MRCHeader CDFAlign::getMrcHeader(const char *filename)
{
	MRC mrc;
	MRCZ mrcz;
	MRCT mrct;
	MRCHeader header;
	memset(&header,0,sizeof(MRCHeader));
	if(isMRC(filename) && mrc.open(filename,"rb") > 0)
	{
		mrc.getHeader(&header);;
		mrc.close();
		return header;
	}
	else if (isMRCZ(filename) && mrcz.open(filename,"rb") > 0) 
        {
                mrcz.getHeader(&header);;
                mrcz.close();
                return header;
        }
	else if (isTIFF(filename) && mrct.open(filename,"rb") > 0) 
        {
                mrct.getHeader(&header);;
                mrct.close();
                return header;
        }
	return header;
}

void CDFAlign::RunAlign()
{
	// TODO: Add your control notification handler code here
	//UpdateData(true);
	
	m_tids.clear();
	
	pthread_t tid;
	int terror;
	terror=pthread_create(&tid,NULL,ThreadFunc_cuAlign,(void *)this);
  	if(terror!=0)
  	{
		TextOutput("Error: Failed to create pthread: Align\n");
		return;
   }
   m_tids.push_back(tid);
   
   //wait for finish
   void *TReturn;
   int i;
	for(i=0;i<m_tids.size();i++)
	{
   	terror=pthread_join(m_tids[i],&TReturn);
   	if(terror!=0)
   	{
      	TextOutput("Warnning: Thread doesn't exit. Something may be wrong.\n");
   	}
   	
   }
   m_tids.clear();
}


void* CDFAlign::ThreadFunc_cuAlign(void* p)
{
	CDFAlign *pThis=(CDFAlign *)p;
	APARA &para=pThis->m_para;
	pThis->m_bRun=true;

	char str[512];
	int j;

	//open stack file
	//test file type
	//
	MRC *  stack;
	
	if (isTIFF(pThis->m_fnStack, 1))
		stack = new MRCT(pThis->m_fnStack,"rb");
	else if (isMRCZ(pThis->m_fnStack, 1))
		stack = new MRCZ(pThis->m_fnStack,"rb");
	else if (isMRC(pThis->m_fnStack, 1))
		stack = new MRC(pThis->m_fnStack,"rb");
	else 
	{
                Message("Only MRC, MRC-gz, TIFF file formats are supported.");
                pThis->m_bRun=false;
                return (void *)0;
        }


	if (stack == NULL)
	{
		sprintf(str,"Error: Failed to open stack %s .",pThis->m_fnStack);
		Message(str);
		pThis->m_bRun=false;
		return (void *)0;
	}

	//get image size
	int nx=stack->getNx();
	int ny=stack->getNy();
	int nz=stack->getNz();
	sprintf(str,"Input Stack: Nx(%d) Ny(%d) Nz(%d) Mode(%d)\n\n",nx,ny,nz,stack->getMode());
	pThis->TextOutput(str);

	
	float bin=para.bin;
	if(bin<=0) return (void *)0;

	float xscale=1.0/para.xscale;
	float yscale=1.0/para.yscale;

	int offsetx=para.crop_offsetx;
	int offsety=para.crop_offsety;
	int nsamUnbin=verifyCropSize(nx,ny, offsetx, offsety, para.crop_nsam,bin);
	if(nsamUnbin<=0)
	{
		Message("Error: Wrong image Size.");
		pThis->m_bRun=false;
		return (void *)0;
	}
	int nsam=(int)((float)nsamUnbin/bin)/2*2;
	int nsamb=nsam+2;
//	if(bin==1) sprintf(str,"Crop Image: Offset(%d %d) Dim(%d)\n",offsetx,offsety,nsamUnbin);
//	else sprintf(str,"Crop Image: Offset(%d %d) RawDim(%d) BinnedDim(%d)\n",offsetx,offsety,nsamUnbin,nsam);
	sprintf(str,"Crop Image: Offset(%d %d) RawDim(%d) BinnedDim(%d)\n",offsetx,offsety,nsamUnbin,nsam);
	pThis->TextOutput(str);
	pThis->m_nsam=nsam;
	pThis->m_nsamRaw=(int)((float)nx/bin);

	//allocate memeory	
	size_t size=nsam*nsam;
	size_t sizeb=nsamb*nsam;
	int sizebUnbin=(nsamUnbin+2)*nsamUnbin;
	if(para.nStart<0) para.nStart=0;
	if(para.nEnd>=nz) para.nEnd=nz-1;
	int nframe=para.nEnd-para.nStart+1;
	pThis->UpdateDisplay();

	//host memory
	float *bufmrc=new float[nx*ny];
	float *bufmrc2=new float[nx*ny]; // dark/gain corrected
//	float *bufscale=new float[nx*ny]; // temp buffer for image stretching
//	float *bufscale2=new float[nx*ny]; // temp buffer for image stretching
	float *bufmrcfft=new float[sizebUnbin];
	float *bufdark=new float[nx*ny];
	float *bufnorm=new float[nx*ny];
	float *bufdark2=new float[nx*ny];
	float *bufnorm2=new float[nx*ny];

	float *htmp=new float[sizeb];
	float *hbuf=new float[sizeb*nframe];  //host memory for entir stack
	float *hbuf2=new float[sizeb*nframe];  //host memory for entire stack, dark/gain corrected
	float *hdisp=new float[sizeb];
	float *hFSCRaw0=new float[sizeb];  //even number
	float *hFSCRaw1=new float[sizeb];  //odd
	float *hFSCCorr0=new float[sizeb];  //even number
	float *hFSCCorr1=new float[sizeb];   //odd
	sprintf(str,"Allocate host memory: %f Gb\n",(6*nx*ny+sizeb*(2*nframe+6)+sizebUnbin)/256.0/1024.0/1024.0);
	pThis->TextOutput(str);
	if(hbuf==0 || hbuf2==0)
	{
		if(bufmrc!=NULL) delete [] bufmrc;
		if(bufmrc2!=NULL) delete [] bufmrc2;
		Message("Failed to allocate host memeory.");
		pThis->m_bRun=false;
		return (void *)0;
	}


	//device memory
	bool success=initGPU(para.GPUNum);
	if(!success)
	{
		sprintf(str,"Failed to initialize GPU #%d.",para.GPUNum);
		Message(str);
		delete [] bufmrc;
		delete [] hbuf;
		delete [] bufmrc2;
//		delete [] bufscale;
//		delete [] bufscale2;
		delete [] hbuf2;
		delete [] bufdark;
		delete [] bufnorm;
		delete [] bufdark2;
		delete [] bufnorm2;

		pThis->m_bRun=false;
		return (void *)0;
	}
	

	float *dsum=0;
	float *dsumcorr=0;
	float *dfft=0;
	float *dtmp=0;
	GPUMemAlloc((void **)&dsum,sizeof(float)*sizeb);	
	GPUMemAlloc((void **)&dsumcorr,sizeof(float)*sizeb);	
	GPUMemAlloc((void **)&dtmp,sizeof(float)*sizeb);
	hipfftHandle fft_plan,ifft_plan;
	
	//prepare fft for unbinned image
	fft_plan=GPUFFTPlan(nsamUnbin);
	GPUSync();
	GPUMemAlloc((void **)&dfft,sizeof(float)*sizebUnbin);


	//make a list 
	int sizec=(nsam/2+1)*nsam;
	MASK *hPosList=new MASK[sizec];
	MASK *dPosList=0;
	MkPosList(hPosList,nsam,para.bfactor);
	GPUMemAlloc((void **)&dPosList,sizeof(MASK)*sizec);
	GPUMemH2D((void **)dPosList,(void **)hPosList,sizeof(MASK)*sizec);

	size_t theFree, theTotal;
	GPUMemCheck(theFree,theTotal);
	sprintf(str,"GPU memory:  free:%.0fMb    total:%.0fMb\n", theFree/1024.0/1024.0, theTotal/1024.0/1024.0);
	pThis->TextOutput(str);

//Read dark/gain reference
	MRC darkref;
	if(strlen(pThis->m_fnDark))
	{
		if(darkref.open(pThis->m_fnDark,"rb")<=0)
		{
			sprintf(str,"Error: Failed to open dark reference %s .",pThis->m_fnDark);
			Message(str);
			pThis->m_bRun=false;
			return (void *)0;
		}
		if (nx!=darkref.getNx() || ny!=darkref.getNy())
		{
			sprintf(str,"Error: Image dimension of dark reference %s differs from image stack .",pThis->m_fnDark);
			Message(str);
			pThis->m_bRun=false;
			return (void *)0;
		}
		if(darkref.read2DIm_32bit(bufdark,0)!=darkref.getImSize())
		{
			sprintf(str,"Error: Failed to read dark reference %s .",pThis->m_fnDark);
			Message(str);
			pThis->m_bRun=false;
			return (void *)0;
		}
		if (darkref.getNz() > 1)
		{
			Message("Processing two dark references.");
			darkref.read2DIm_32bit(bufdark2,1);
		}
		else
		{
			Message("Processing single dark reference.");
			//darkref.read2DIm_32bit(bufdark2,0);
			for(int k=0;k<nx*ny;k++)
			{
				bufdark2[k] = 0.0;
			}
		}
		
	}
	MRC normref;
	if(strlen(pThis->m_fnNorm))
	{
		if(normref.open(pThis->m_fnNorm,"rb")<=0)
		{
			sprintf(str,"Error: Failed to open gain reference %s .",pThis->m_fnNorm);
			Message(str);
			pThis->m_bRun=false;
			return (void *)0;
		}
		if (nx!=normref.getNx() || ny!=normref.getNy())
		{
			sprintf(str,"Error: Image dimension of gain reference %s differs from image stack .",pThis->m_fnNorm);
			Message(str);
			pThis->m_bRun=false;
			return (void *)0;
		}
		if(normref.read2DIm_32bit(bufnorm,0)!=normref.getImSize())
		{
			sprintf(str,"Error: Failed to read gain reference %s .",pThis->m_fnNorm);
			Message(str);
			pThis->m_bRun=false;
			return (void *)0;
		}
		if (normref.getNz() > 1)
		{
			Message("Processing two norm references.");
			normref.read2DIm_32bit(bufnorm2,1);
		}
		else
		{
			Message("Processing single norm reference.");
			//normref.read2DIm_32bit(bufnorm2,0);
			for(int k=0;k<nx*ny;k++)
			{
				bufnorm2[k] = 1.0;
			}
		}

	}
//End reading dark/gain reference
	//Read stack
	pThis->TextOutput("\nRead stack:\n");
	
	float sx=0;
	float sy=0;
	float shiftx,shifty,cc;
	float avgcc=0.0;
	bool bFSCEven=true;

	//1. calculate sum
	GPUMemZero((void **)&dsum,sizeof(float)*sizeb);
	GPUSync();
	GPUMemZero((void **)&dsumcorr,sizeof(float)*sizeb);
	GPUSync();
	for(j=para.nStart;j<=para.nEnd;j++)
	{
		//read from file and crop
		if(stack->read2DIm_32bit(bufmrc,j)!=stack->getImSize())
		{
			sprintf(str,"Error when reading #%03d\n",j);
			pThis->TextOutput(str);
		}
		// Do dark/gain correction
		if(strlen(pThis->m_fnNorm) && strlen(pThis->m_fnDark)) 
		{
			pThis->TextOutput("Correcting dark/gain...\n");
			for(int k=0;k<nx*ny;k++)
			{
				bufmrc2[k]=(bufmrc[k]-bufdark2[k])*bufnorm2[k];
				bufmrc[k]=(bufmrc[k]-bufdark[k])*bufnorm[k];
			}
		}
		else if (strlen(pThis->m_fnNorm))
		{
			pThis->TextOutput("Correcting gain...\n");
			for(int k=0;k<nx*ny;k++)
			{
				bufmrc2[k]=bufmrc[k]*bufnorm2[k];
				bufmrc[k]=bufmrc[k]*bufnorm[k];
			}
		}
		else if (strlen(pThis->m_fnDark))
		{
			pThis->TextOutput("Correcting dark...\n");
			for(int k=0;k<nx*ny;k++)
			{
				bufmrc2[k]=bufmrc[k]-bufdark2[k];
				bufmrc[k]=bufmrc[k]-bufdark[k];
			}
		}
		else
		{
//			for(int k=0;k<nx*ny;k++) bufmrc2[k]=bufmrc[k];
			memcpy(bufmrc2, bufmrc, sizeof(float)*nx*ny);
		}
// Stretch image
		if (xscale<1.0 || yscale<1.0)
		{
			int addr_orig, addr_new;
			for (int ky=ny-1; ky>0; ky--)
			{
				for (int kx=nx-1; kx>0; kx--)
				{
					// Nearest neighbor interpolation.
					addr_orig = int(ky*yscale+0.5)*nx+int(kx*xscale+0.5);
					addr_new = ky*nx+kx;
					bufmrc[addr_new]=bufmrc[addr_orig];
					bufmrc2[addr_new]=bufmrc2[addr_orig];
				}
			}
		}
// FFT the dark/gain corrected image.
		crop2fft(bufmrc2,nx,ny,bufmrcfft,offsetx,offsety,nsamUnbin,bin);
		//copy to GPU
		GPUMemH2D((void *)dfft,(void *)bufmrcfft,sizeof(float)*sizebUnbin);
		//do fft
		GPUFFT2d(dfft,fft_plan);
		GPUSync();
		//do binning
		if(bin>1.0001)
		{
			GPUMemBinD2D(dtmp, dfft, nsam, nsamUnbin);
			GPUMemD2D(dfft, dtmp, sizeof(float)*sizeb);
		}
		//Sum
//		if(j>=para.nStartSum && j<=para.nEndSum)
//		{
//			if(bFSCEven) GPUAdd(dsum,dfft,sizeb);
//			else GPUAdd(dsumcorr,dfft,sizeb);
//			bFSCEven=!bFSCEven;
//		}
		//copy ffted image to host
		GPUMemD2H((void *)(hbuf2+(j-para.nStart)*sizeb),(void *)dfft,sizeof(float)*sizeb);
		GPUSync();
// 

		crop2fft(bufmrc,nx,ny,bufmrcfft,offsetx,offsety,nsamUnbin,bin);
		
		//copy to GPU
		GPUMemH2D((void *)dfft,(void *)bufmrcfft,sizeof(float)*sizebUnbin);
		//do fft
		GPUFFT2d(dfft,fft_plan);
		GPUSync();

		//do binning
		if(bin>1.0001)
		{
			GPUMemBinD2D(dtmp, dfft, nsam, nsamUnbin);
			GPUMemD2D(dfft, dtmp, sizeof(float)*sizeb);
		}

		//Sum
		if(j>=para.nStartSum && j<=para.nEndSum)
		{
			if(bFSCEven) GPUAdd(dsum,dfft,sizeb);
			else GPUAdd(dsumcorr,dfft,sizeb);
			bFSCEven=!bFSCEven;
		}
		//copy ffted image to host
		GPUMemD2H((void *)(hbuf+(j-para.nStart)*sizeb),(void *)dfft,sizeof(float)*sizeb);
		GPUSync();

		sprintf(str,"......Read and sum frame #%03d   mean:%f\n",j,(hbuf+(j-para.nStart)*sizeb)[0]/nsam/nsam);
		pThis->TextOutput(str);
	}
	GPUMemD2H((void *)hFSCRaw0,(void *)dsum,sizeof(float)*sizeb);
	GPUMemD2H((void *)hFSCRaw1,(void *)dsumcorr,sizeof(float)*sizeb);
	GPUAdd(dsum,dsumcorr,sizeb);
	GPUSync();
	

	//free memory for unbined image
	delete [] bufmrcfft;
	delete stack;
	bufmrcfft=0;
	GPUMemFree((void **)&dfft);
	GPUFFTDestroy(fft_plan);
	fft_plan=0;	
	//finish GPU memory allocate
	GPUMemAlloc((void **)&dfft,sizeof(float)*sizeb);
	GPUMemZero((void **)&dsumcorr,sizeof(float)*sizeb);
	GPUSync();
	ifft_plan=GPUIFFTPlan(nsam);
	GPUSync();


	//Make fft modulus for display
	if(para.bDispFFTRaw)
	{
		GPUSync();
		GPUFFTLogModulus(dfft, dsum, nsam, para.fftscale);
		GPUSync();
		GPUMemD2H((void *)hdisp,(void *)dfft,sizeof(float)*(nsam/2+1)*nsam);
		//pThis->FFTOutputRaw(hdisp);   //has been move to below
	}
	//copy sum image to host for save and display
	if(para.bDispFFTRaw || para.bSaveRawSum)
	{
		GPUIFFT2d(dsum,ifft_plan);
		GPUSync();
		GPUMultiplyNum(dsum,1.0/size,sizeb);
		GPUMemD2H((void *)htmp,(void *)dsum,sizeof(float)*sizeb);
		fft2buf(bufmrc,htmp,nsam);
	}
	//save
	MRC mrcraw;
	if(para.bSaveRawSum)
	{
		//write to file
		mrcraw.open(pThis->m_fnRawsum,"wb");
		mrcraw.createMRC(bufmrc,nsam,nsam,1);
		//stats
		sprintf(str,"Mean=%f   Min=%f   Max=%f\n",mrcraw.m_header.dmean,mrcraw.m_header.dmin,mrcraw.m_header.dmax);
		pThis->TextOutput(str);
		mrcraw.close();
		sprintf(str,"Save Unaligned Sum to: %s\n",pThis->m_fnRawsum);
		pThis->TextOutput(str);
	}
	

	//2. frame to frame shift
	pThis->TextOutput("\nCalculate relative drift between frames\n");
	Matrix<complex<double> > A;
	vector<complex<int> > compList;
	int ncomp=OD_SetEquation_All(A,compList, nframe, para.FrameDistOffset);
	Vector<complex<double> > b=Vector<complex<double> >(ncomp);
	int box=para.CCPeakSearchDim;
	float *hboxmap=new float[box*box*ncomp];
	int par0,par1;
	for(j=0;j<ncomp;j++)
	{
		par0=compList[j].real();
		par1=compList[j].imag();
		//copy to GPU
		GPUMemH2D((void *)dsum,(void *)(hbuf+par0*sizeb),sizeof(float)*sizeb);
		GPUMemH2D((void *)dfft,(void *)(hbuf2+par1*sizeb),sizeof(float)*sizeb);
		//shift and cc
		sx=0;
		sy=0;
		GPUShiftCC(dfft, dsum, dPosList,sx, sy, nsam);
		GPUSync();
		//do ifft
		GPUIFFT2d(dfft,ifft_plan);
		GPUSync();
		//find shift
		cc=FindShift(dfft,nsam, hboxmap+j*box*box, box, shiftx, shifty, para.NoisePeakSize-1);
		b[j]=complex<double>(shiftx,shifty);
		avgcc+=cc;
		sprintf(str,"......%03d Frame #%03d VS #%03d xy-shift: %8.4f %8.4f      CC:%f\n",j,par0+para.nStart,par1+para.nStart,shiftx,shifty,cc);
		pThis->TextOutput(str);
	}

	//display the RawImageFFT here due to FFTW thread safety issue
	if(para.bDispFFTRaw)
	{
		pThis->FFTOutputRaw(hdisp);
	}


	//3. sovle overdetermined equation
	Vector<complex<double> > shift=lsSolver(A,b);
	Vector<double> ki=abs(A*shift-b);
	sprintf(str,"\n......ki: First round \n");
	pThis->TextOutput(str);
	for(j=0;j<ki.size();j++)
	{
		par0=compList[j].real();
		par1=compList[j].imag();
		sprintf(str,"......ki #%03d of Frame #%03d VS #%03d: %8.4lf \n",j+para.nStart,par0+para.nStart,par1+para.nStart,ki[j]);
		pThis->TextOutput(str);
	}
	sprintf(str,"................................Average ki: %8.4lf \n\n",sum(ki)/ki.size());
	pThis->TextOutput(str);
	//display CCMap
	if(para.bDispCCMap)
	{
		pThis->CCMapOutput(hboxmap,(void *)&ki);
	}
	//3.1 re-sovle overdetermined equation after removing large ki elments
	double kiThresh=para.kiThresh;
	vector<int> goodlist=OD_Threshold(A, b, ki, kiThresh);
	shift=lsSolver(A,b);
	ki=abs(A*shift-b);
	sprintf(str,"......ki: Second round \n");
	pThis->TextOutput(str);
	for(j=0;j<ki.size();j++)
	{
		par0=compList[goodlist[j] ].real();
		par1=compList[goodlist[j] ].imag();
		sprintf(str,"......ki #%03d of Frame #%03d VS #%03d: %8.4f \n",j+para.nStart,par0+para.nStart,par1+para.nStart,ki[j]);
		pThis->TextOutput(str);
	}
	sprintf(str,"................................Average ki: %8.4lf \n\n",sum(ki)/ki.size());
	pThis->TextOutput(str);

	//4. Do the iterative alignment
	//
	pThis->TextOutput("Begin Iterative Alignment... \n");
	bool converged=false;
	complex<double> zeroshift=0;
	complex<double> allshift=0;
	vector<complex<double> > newshift;
	newshift.push_back(zeroshift);
	for (j=0; j<shift.size();j++)
	{
		newshift.push_back(allshift+shift[j]);
		allshift+=shift[j];
	}

	vector<complex<double> > oldshift;
	oldshift=newshift;
	
	double rmsd=0;
	for (j=0; (j<pThis->m_iter) && (! converged); j++)
	{
		sprintf(str,"Iteration [%d/%d]\n", j, pThis->m_iter);
		pThis->TextOutput(str);
		//4.1 Do the sum
		//
		GPUMemH2D((void *)dsum,(void *)(hbuf),sizeof(float)*sizeb);
		GPUShift(dsum,dPosList,-oldshift[0].real(),-oldshift[0].imag(), nsam);
		sprintf(str,"......Add Frame #%03d with xy shift: %8.4lf %8.4lf\n",para.nStart,-oldshift[0].real(),-oldshift[0].imag());
		pThis->TextOutput(str);
		for (int k=para.nStart+1; k<=para.nEnd; k++)
		{
			int position=k-para.nStart;
			GPUMemH2D((void *)dfft,(void *)(hbuf+position*sizeb),sizeof(float)*sizeb);
			GPUSync();
			GPUShift(dfft,dPosList,-oldshift[position].real(),-oldshift[position].imag(), nsam);
			GPUSync();
			GPUAdd(dsum, dfft, sizeb);
			sprintf(str,"......Add Frame #%03d with xy shift: %8.4lf %8.4lf\n",k,-oldshift[position].real(),-oldshift[position].imag());
			pThis->TextOutput(str);
			GPUSync();
		}
			
		pThis->TextOutput("Sum complete... \n");

		//4.2 align to the sum
		//
		float *hboxmap2=new float[box*box*nframe];
		avgcc=0;
		for (int k=para.nStart; k<=para.nEnd; k++)
		{
			int position=k-para.nStart;
			GPUMemH2D((void *)dfft,(void *)(hbuf+position*sizeb),sizeof(float)*sizeb);
			GPUSync();
	//		sx=oldshift[position].real();
	//		sy=oldshift[position].imag();
			sx=0;
			sy=0;
			GPUShiftCC(dfft, dsum, dPosList,sx, sy, nsam);
			GPUSync();
			GPUIFFT2d(dfft,ifft_plan);
			GPUSync();
			cc=FindShift(dfft,nsam, hboxmap2+k*box*box, box, shiftx, shifty, para.NoisePeakSize-1);
			avgcc+=cc;
			newshift[position]=complex<double>(shiftx,shifty);
			complex<double> diffshift=oldshift[position]-newshift[position];
			sprintf(str,"......%03d Frame #%03d VS <sum> xy-shift: %8.4f %8.4f (%8.4f %8.4f) CC:%f\n",position,k,shiftx,shifty,diffshift.real(),diffshift.imag(),cc);
			pThis->TextOutput(str);
		}

		if (para.bSaveCCmap) 
		{
			sprintf(str, "iterCC%03d.mrc", j);
			buf2mrc(str,hboxmap2,box,box,(para.nStart-para.nEnd+1));
			sprintf(str,"Save CC map to: %s\n", str);
			pThis->TextOutput(str);
		}

		delete [] hboxmap2;
		
		//4.3 update shift and track changes
		//
		complex<double> refshift=newshift[0];
		for (int k=para.nStart; k<=para.nEnd; k++)
		{
			int position=k-para.nStart;
			complex<double> diffshift=oldshift[position]-newshift[position];
			oldshift[position]=newshift[position]-refshift;
			rmsd+=diffshift.real()*diffshift.real()+diffshift.imag()*diffshift.imag();
		}
		
		sprintf(str,"Average CC : %8.4f\n", avgcc/nframe);
		pThis->TextOutput(str);
		rmsd/=nframe;
		if (rmsd < pThis->m_iterthres*pThis->m_iterthres) 
		{
			converged=true;
			sprintf(str,"Converged at iteration [%d/%d]\n", j, pThis->m_iter);
			pThis->TextOutput(str);
		}
			
		sprintf(str,"RMSD to <sum> : %8.4f  (%8.4f for convergence)\n", sqrt(rmsd), pThis->m_iterthres);
		pThis->TextOutput(str);

	}
	sprintf(str, "Final RMSD to <sum> : %8.4f at iteration [%d/%d]\n\nShifts (delta) after iterative aligment\n", sqrt(rmsd), j-1, pThis->m_iter);
	pThis->TextOutput(str);

	//Convert oldshift to shift
	//
	sprintf(str, "...Frame #%03d xy-shift %8.4f %8.4f (%8.4f %8.4f)\n", 0,0,0,0,0);
	pThis->TextOutput(str);
	allshift=0;
	for (int k=0; k<shift.size(); k++)
	{
		allshift+=shift[k];
		complex<double> tempshift=0;
		tempshift=oldshift[k+1]-oldshift[k];
		complex<double> tempdiffshift=0;
		tempdiffshift=allshift-oldshift[k+1];
		sprintf(str, "...Frame #%03d xy-shift %8.4f %8.4f (%8.4f %8.4f)\n", k+1, oldshift[k+1].real(), oldshift[k+1].imag(), tempdiffshift.real(), tempdiffshift.imag());
		pThis->TextOutput(str);
		shift[k]=oldshift[k+1]-oldshift[k];
	}


	//output final shift
	pThis->TextOutput("Final shift:\n");
	vector<complex<double> > shiftlist;
	complex<double> totalshift=0;
	sprintf(str,"......Shift of Frame #%03d : %8.4f %8.4f\n",para.nStart,totalshift.real(),totalshift.imag());
//	sprintf(str,"......Shift of Frame #%03d : %8.4f %8.4f\n",para.nStart,oldshift[0].real(),oldshift[0].imag());
	pThis->TextOutput(str);
	shiftlist.push_back(totalshift);
//	shiftlist.push_back(oldshift[0]);
	for(j=0;j<shift.size();j++)
	{
		totalshift=totalshift+shift[j];
		sprintf(str,"......Shift of Frame #%03d : %8.4f %8.4f\n",j+para.nStart+1,totalshift.real(),totalshift.imag());
//		sprintf(str,"......Shift of Frame #%03d : %8.4f %8.4f\n",j+para.nStart+1,oldshift[j+1].real(),oldshift[j+1].imag());
		pThis->TextOutput(str);
		shiftlist.push_back(totalshift);
//		shiftlist.push_back(oldshift[j+1]);
	}
	pThis->PlotOutput(shiftlist);

	//save CCMap image
	if(para.bSaveCCmap) 
	{
		buf2mrc(pThis->m_fnCCmap,hboxmap,box,box,ncomp);
		sprintf(str,"Save CC map to: %s\n",pThis->m_fnCCmap);
		pThis->TextOutput(str);
	}
		

	
	MRC stackCorr;
	if(para.bSaveStackCorr)
	{
		stackCorr.open(pThis->m_fnStackCorr,"wb");
		stackCorr.m_header.nx=nsam;
		stackCorr.m_header.ny=nsam;
		stackCorr.m_header.nz=para.nEndSum-para.nStartSum+1;
		stackCorr.updateHeader();
	}

	//3. correct xy-shift
	int nStartSum=para.nStartSum-para.nStart;
	int nEndSum=para.nEndSum-para.nStart;
	int nEndSum2=para.nEndSum2-para.nStart;

	if(nStartSum<=0) nStartSum=0;
	if(nEndSum<=nStartSum || nEndSum>=nframe) nEndSum=nframe-1;
	if(nEndSum2<=nStartSum || nEndSum2>=nframe) nEndSum2=nframe-1;
	if(nEndSum2>nEndSum) nEndSum2=nEndSum;

	sprintf(str,"\nSum Frame #%03d - #%03d\n",nStartSum+para.nStart,nEndSum+para.nStart);
	pThis->TextOutput(str);
	//reset memory
	GPUMemZero((void **)&dsum,sizeof(float)*sizeb);
	GPUSync();
	GPUMemZero((void **)&dsumcorr,sizeof(float)*sizeb);
	GPUSync();
	//calculate middle frame shift
	complex<double> midshift=0.0;
	int RefFrame=nz/2+1;
	if(para.bAlignToMid) 
	{
		if(RefFrame<para.nStart) RefFrame=para.nStart;
		if(para.nStartSum>para.nEnd) para.nStartSum=para.nEnd;
		for(j=0;j<RefFrame-para.nStart;j++) midshift+=shift[j];
	}

	//Add(copy) first frame to GPU
	totalshift=0;
	for(j=1;j<nStartSum+1;j++)
	{
		totalshift+=shift[j-1];
	}
	GPUMemH2D((void *)dsumcorr,(void *)(hbuf+nStartSum*sizeb),sizeof(float)*sizeb);
	if(para.bAlignToMid) GPUShift(dsumcorr,dPosList,-totalshift.real()+midshift.real(),-totalshift.imag()+midshift.imag(), nsam);
	GPUSync();
	bFSCEven=false;
	sprintf(str,"......Add Frame #%03d with xy shift: %8.4lf %8.4lf\n",nStartSum+para.nStart,-totalshift.real()+midshift.real(),-totalshift.imag()+midshift.imag());
	pThis->TextOutput(str);
	//Save stack
	if(para.bSaveStackCorr)
	{
		GPUMemD2D((void *)dfft,(void *)dsumcorr,sizeof(float)*sizeb);
		GPUIFFT2d(dfft,ifft_plan);
		GPUSync();
		GPUMultiplyNum(dfft,1.0/size,sizeb);
		GPUSync();
		GPUMemD2H((void *)htmp,(void *)dfft,sizeof(float)*sizeb);
		fft2buf(bufmrc,htmp,nsam);
		stackCorr.write2DIm(bufmrc,0);
	}
	//*******
	//sum other frame
	for(j=nStartSum+1;j<=nEndSum2;j++)
	{
		totalshift+=shift[j-1];
		
		//copy to GPU
		GPUMemH2D((void *)dfft,(void *)(hbuf+j*sizeb),sizeof(float)*sizeb);
		//shift
		GPUShift(dfft,dPosList,-totalshift.real()+midshift.real(),-totalshift.imag()+midshift.imag(), nsam);
		GPUSync();
		//Sum
		if(bFSCEven) GPUAdd(dsumcorr,dfft,sizeb);
		else GPUAdd(dsum,dfft,sizeb);
		bFSCEven=!bFSCEven;

		sprintf(str,"......Add Frame #%03d with xy shift: %8.4lf %8.4lf\n",j+para.nStart,-totalshift.real()+midshift.real(),-totalshift.imag()+midshift.imag());
		pThis->TextOutput(str);

		//save stack
		if(para.bSaveStackCorr)
		{
			GPUIFFT2d(dfft,ifft_plan);
			GPUSync();
			GPUMultiplyNum(dfft,1.0/size,sizeb);
			GPUSync();
			GPUMemD2H((void *)htmp,(void *)dfft,sizeof(float)*sizeb);
			fft2buf(bufmrc,htmp,nsam);
			stackCorr.write2DIm(bufmrc,j-nStartSum);
		}
	}

//output the second sum image (less frames).
	if (para.nEndSum2>0 && para.nEndSum2<para.nEndSum)
	{
		GPUMemZero((void **)&dtmp,sizeof(float)*sizeb);
		GPUMemD2D(dtmp, dsumcorr, sizeof(float)*sizeb);
		GPUAdd(dtmp,dsum,sizeb);
		GPUSync();
		//copy sum image to host
		float *tsum=dtmp;
		GPUIFFT2d(tsum,ifft_plan);
		GPUMultiplyNum(tsum,1.0/size,sizeb);
		GPUMemD2H((void *)htmp,(void *)tsum,sizeof(float)*sizeb);
		fft2buf(bufmrc,htmp,nsam);
		//save
		char tempname[512]="";
		char subfilename[512]="";
		MRC mrc;
		if(para.bSaveSubAreaCorrSum) 
		{
			strncat(tempname,pThis->m_fnAlignsum2,strlen(pThis->m_fnAlignsum2)-4);
			sprintf(subfilename,"%s_ox%d_oy%d_dim%d.mrc",tempname,offsetx,offsety,nsamUnbin);
			mrc.open(subfilename,"wb");
		}
		else mrc.open(pThis->m_fnAlignsum2,"wb");
		mrc.createMRC(bufmrc,nsam,nsam,1);
		//stats
		sprintf(str,"Mean=%f   Min=%f   Max=%f\n",mrc.m_header.dmean,mrc.m_header.dmin,mrc.m_header.dmax);
		pThis->TextOutput(str);
		mrc.close();
		if(para.bSaveSubAreaCorrSum) sprintf(str,"Save Sum to: %s\n",subfilename);
		else sprintf(str,"Save Sum to: %s\n",pThis->m_fnAlignsum2);
		pThis->TextOutput(str);
	}

//add the rest of the frames
	for(j=nEndSum2+1;j<=nEndSum;j++)
	{
		totalshift+=shift[j-1];
		
		//copy to GPU
		GPUMemH2D((void *)dfft,(void *)(hbuf+j*sizeb),sizeof(float)*sizeb);
		//shift
		GPUShift(dfft,dPosList,-totalshift.real()+midshift.real(),-totalshift.imag()+midshift.imag(), nsam);
		GPUSync();
		//Sum
		if(bFSCEven) GPUAdd(dsumcorr,dfft,sizeb);
		else GPUAdd(dsum,dfft,sizeb);
		bFSCEven=!bFSCEven;

		sprintf(str,"......Add Frame #%03d with xy shift: %8.4lf %8.4lf\n",j+para.nStart,-totalshift.real()+midshift.real(),-totalshift.imag()+midshift.imag());
		pThis->TextOutput(str);

		//save stack
		if(para.bSaveStackCorr)
		{
			GPUIFFT2d(dfft,ifft_plan);
			GPUSync();
			GPUMultiplyNum(dfft,1.0/size,sizeb);
			GPUSync();
			GPUMemD2H((void *)htmp,(void *)dfft,sizeof(float)*sizeb);
			fft2buf(bufmrc,htmp,nsam);
			stackCorr.write2DIm(bufmrc,j-nStartSum);
		}
	}

	//close save stack
	if(para.bSaveStackCorr) stackCorr.close();
	//final sum
	GPUMemD2H((void *)hFSCCorr0,(void *)dsumcorr,sizeof(float)*sizeb);
	GPUMemD2H((void *)hFSCCorr1,(void *)dsum,sizeof(float)*sizeb);
	GPUAdd(dsumcorr,dsum,sizeb);
	GPUSync();

	//Make fft modulus for display
	if(para.bDispFFTCorr)
	{
		GPUSync();
		GPUFFTLogModulus(dfft, dsumcorr, nsam, para.fftscale);
		GPUSync();
		GPUMemD2H((void *)hdisp,(void *)dfft,sizeof(float)*(nsam/2+1)*nsam);
		pThis->FFTOutputCorr(hdisp);   //has been move to below
	}

	//copy binned sum to display
	if(para.bDispSumCorr)
	{
		GPUMemBinD2H(hdisp, dsumcorr, DISPDIM, nsam);
		pThis->ImageOutput(hdisp);
	}

	//copy sum image to host
	float *tsum=dsumcorr;
	GPUIFFT2d(tsum,ifft_plan);
	GPUMultiplyNum(tsum,1.0/size,sizeb);
	GPUMemD2H((void *)htmp,(void *)tsum,sizeof(float)*sizeb);
	fft2buf(bufmrc,htmp,nsam);
	
	//save
	char tempname[512]="";
	char subfilename[512]="";
	MRC mrc;
	if(para.bSaveSubAreaCorrSum) 
	{
		strncat(tempname,pThis->m_fnAlignsum,strlen(pThis->m_fnAlignsum)-4);
		sprintf(subfilename,"%s_ox%d_oy%d_dim%d.mrc",tempname,offsetx,offsety,nsamUnbin);
		mrc.open(subfilename,"wb");
	}
	else mrc.open(pThis->m_fnAlignsum,"wb");
	mrc.createMRC(bufmrc,nsam,nsam,1);
	//stats
	sprintf(str,"Mean=%f   Min=%f   Max=%f\n",mrc.m_header.dmean,mrc.m_header.dmin,mrc.m_header.dmax);
	pThis->TextOutput(str);
	mrc.close();
	if(para.bSaveSubAreaCorrSum) sprintf(str,"Save Sum to: %s\n",subfilename);
	else sprintf(str,"Save Sum to: %s\n",pThis->m_fnAlignsum);
	pThis->TextOutput(str);

	if(para.bLogFSC)
	{
		pThis->PlotFSC((hipComplex *)hFSCRaw0, (hipComplex *)hFSCRaw1, (hipComplex *)hFSCCorr0, 
					(hipComplex *)hFSCCorr1,hPosList,nsam,totalshift);
	}

	sprintf(str,"Done.\n");
	pThis->TextOutput(str);

	delete [] bufmrc;
	delete [] hbuf;
	delete [] bufmrc2;
//	delete [] bufscale;
//	delete [] bufscale2;
	delete [] hbuf2;
	delete [] hPosList;
	delete [] bufdark;
	delete [] bufnorm;
	delete [] bufdark2;
	delete [] bufnorm2;

	GPUMemFree((void **)&dPosList);
	GPUMemFree((void **)&dsum);
	GPUMemFree((void **)&dsumcorr);
	GPUMemFree((void **)&dfft);
	GPUMemFree((void **)&dtmp);
	//GPUFFTDestroy(fft_plan);
	GPUFFTDestroy(ifft_plan);

	delete [] htmp;
	delete [] hboxmap;
	delete [] hdisp;
	delete [] hFSCRaw0;
	delete [] hFSCRaw1;
	delete [] hFSCCorr0;
	delete [] hFSCCorr1;

	ResetGPU();
	pThis->Done();

	return (void *)0;
}
