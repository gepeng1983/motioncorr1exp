#include "hip/hip_runtime.h"
#include "cufunc.h"
#include <string.h>
#include "mrc.h"
#include "func.h"
#include <signal.h> 
#define PI 3.141592653589793
#define BLOCKSIZE 1024


static __device__ hipComplex conj(hipComplex f)
{
	f.y*=-1.0;
	return f;
}


static __device__ hipComplex cXc(hipComplex a, hipComplex b) // a*b
{
	hipComplex c;
	c.x=a.x*b.x-a.y*b.y;
	c.y=a.x*b.y+a.y*b.x;
	return c;
}

static __device__ float cabs(hipComplex a) // a*b
{
	return sqrt(a.x*a.x+a.y*a.y);
}

bool initGPU(int GPUNum)
{
	//initional CUDA device
	int ngpu;
	hipGetDeviceCount(&ngpu);
	if(ngpu <= 0)
	{
		return false;
	}
	if(GPUNum>=ngpu)
	{
		printf("GPU ID %d is out of range(%d). Abort.\n",GPUNum,ngpu);
		return false;
	}

	hipDeviceProp_t prop;
	if(hipGetDeviceProperties(&prop, GPUNum) == hipSuccess) 
	{
		printf("Use GPU: #%d %s\n",GPUNum,prop.name);
		if(prop.kernelExecTimeoutEnabled)
		{
			printf("Warnning: This GPU is also used for display, may not stable.\n");
		}
	}


	if(hipSetDevice(GPUNum)!=hipSuccess)
	{
		printf("Error: Failed to set CUDA Device #%d. Abort.\n",GPUNum);
		return false;
	}
	
	signal(SIGINT, siginthandler);

	return true;
}

bool ResetGPU()
{
	if(hipDeviceReset()!=hipSuccess) return false;
	return true;
}

void siginthandler(int param) 
{   
	if(ResetGPU()) printf(" GPU was reset successfully after process was killed.\n"); 
	else printf(" Error: Failed to reset GPU.\n"); 
	exit(1); 
}

int getGPUList(vector<string> &namelist)
{
	int ngpu;
	hipGetDeviceCount(&ngpu);
	if(ngpu <= 0)
	{
		return 0;
	}

	namelist.clear();
	int i;
	string str;
	for(i = 0; i < ngpu; i++) 
	{
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) 
		{
			str=prop.name;
			namelist.push_back(str);
		}
	}
	
	return ngpu;
}

void GPUMemCheck(size_t &theFree, size_t &theTotal)
{
	hipMemGetInfo( &theFree, &theTotal );  
	//printf( "CARD returns:  free:%d  total:%d\n", theFree, theTotal);
}

bool GPUMemAlloc(void **buf, int size)
{
	if(hipMalloc((void **)buf,size)!=hipSuccess) return false;
	return true;
}

bool GPUMemZero(void **buf, int size)
{
	if(hipMemset(*buf,0,size)!=hipSuccess) return false;
	return true;
}

bool GPUMemFree(void **buf)
{
	if(hipFree(*buf)!=hipSuccess) return false;
	*buf=0;
	return true;
}

bool GPUMemH2D(void *dst, void *src, int size)
{
	if(hipMemcpy(dst,src,size,hipMemcpyHostToDevice)!=hipSuccess) return false;
	return true;
}

bool GPUMemD2H(void *dst, void *src, int size)
{
	if(hipMemcpy(dst,src,size,hipMemcpyDeviceToHost)!=hipSuccess) return false;
	return true;
}
bool GPUMemD2D(void *dst, void *src, int size)
{
	if(hipMemcpy(dst,src,size,hipMemcpyDeviceToDevice)!=hipSuccess) return false;
	return true;
}

bool GPUMemBinD2H(float *dst, float *src, int dst_nsam, int src_nsam)
{
	int i;
	int size=sizeof(float)*(dst_nsam+2);
	for(i=0;i<dst_nsam/2;i++)
	{
		//up half
		if(hipMemcpy(dst+i*(dst_nsam+2),src+i*(src_nsam+2),size,hipMemcpyDeviceToHost)!=hipSuccess) return false;

		//down half
		if(hipMemcpy(dst+(dst_nsam-1-i)*(dst_nsam+2),src+(src_nsam-1-i)*(src_nsam+2),size,hipMemcpyDeviceToHost)!=hipSuccess) return false;
	}

	return true;
}
bool GPUMemBinD2D(float *dst, float *src, int dst_nsam, int src_nsam)
{
	int i;
	int size=sizeof(float)*(dst_nsam+2);
	for(i=0;i<dst_nsam/2;i++)
	{
		//up half
		if(hipMemcpy(dst+i*(dst_nsam+2),src+i*(src_nsam+2),size,hipMemcpyDeviceToDevice)!=hipSuccess) return false;

		//down half
		if(hipMemcpy(dst+(dst_nsam-1-i)*(dst_nsam+2),src+(src_nsam-1-i)*(src_nsam+2),size,hipMemcpyDeviceToDevice)!=hipSuccess) return false;
	}

	return true;
}

__global__ void cuFFTLogModulus(float *dMod, hipComplex *dfft, int size, float scale)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dMod[id]=log(1+cabs(dfft[id])*scale);
}
void GPUFFTLogModulus(float *dMod, float *dfft, int nsam, float scale)
{
	int size=(nsam/2+1)*nsam;
	cuFFTLogModulus<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dMod,(hipComplex *)dfft,size,scale);
}

__global__ void cuFFTModulus(float *dMod, hipComplex *dfft, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dMod[id]=cabs(dfft[id]);
}
void GPUFFTModulus(float *dMod, float *dfft, int nsam)
{
	int size=(nsam/2+1)*nsam;
	cuFFTModulus<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dMod,(hipComplex *)dfft,size);
}

hipfftHandle GPUFFTPlan(int nsam)
{
	hipfftHandle plan;
	hipfftResult r=hipfftPlan2d(&plan,nsam,nsam,HIPFFT_R2C);
	//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_PADDING);
	return plan;
}
hipfftHandle GPUIFFTPlan(int nsam)
{
	hipfftHandle plan;
	hipfftResult r=hipfftPlan2d(&plan,nsam,nsam,HIPFFT_C2R);
	//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_PADDING);
	return plan;
}

void GPUFFTDestroy(hipfftHandle &plan)
{
	hipfftDestroy(plan);
	plan=0;
}

bool GPUFFT2d(float* dfft, hipfftHandle plan)
{
	if(hipfftExecR2C(plan,(hipfftReal*)dfft,(hipfftComplex *)dfft)!=HIPFFT_SUCCESS) return false;
	return true;
}

bool GPUIFFT2d(float* dfft, hipfftHandle plan)
{
	if(hipfftExecC2R(plan,(hipfftComplex *)dfft,(hipfftReal*)dfft)!=HIPFFT_SUCCESS) return false;
	return true;
}

bool GPUSync()
{
	if(hipDeviceSynchronize()!=hipSuccess) return false;
	if(hipGetLastError()!=hipSuccess) return false;
	return true;
}


__global__ void cuAdd(float *dst, float *src, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dst[id]+=src[id];
}
void GPUAdd(float *dst, float *src, int size)
{
	cuAdd<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dst,src,size);
}

__global__ void cuMultiplyNum(float *dst, float num, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;
	dst[id]*=num;
}
void GPUMultiplyNum(float *dst, float num, int size)
{
	cuMultiplyNum<<<size/BLOCKSIZE+1,BLOCKSIZE>>>(dst,num,size);
}

//int3[x,y,sign]
void MkPosList(int3 *list, int nsam, float inner_r, float outer_r)
{
	int hnsamb=nsam/2+1;
	int hnsam=nsam/2;
	int i,j;
	int count=0;
	int r2;
	int ri2=int(inner_r*inner_r);
	int ro2=int(outer_r*outer_r);
	for(j=0;j<hnsam;j++)
		for(i=0;i<hnsamb;i++)
		{
			list[count].x=i;
			list[count].y=j;

			r2=list[count].x*list[count].x+list[count].y*list[count].y;
			if(r2<ri2 || r2>ro2) list[count].z=0;
			else 
			{
				if((list[count].x+list[count].y)%2==0) list[count].z=1;
				else list[count].z=-1;
			}

			count++;
		}
	for(j=hnsam;j<nsam;j++)
		for(i=0;i<hnsamb;i++)
		{
			list[count].x=i;
			list[count].y=j-nsam;
			r2=list[count].x*list[count].x+list[count].y*list[count].y;
			if(r2<ri2 || r2>ro2) list[count].z=0;
			else 
			{
				if((list[count].x+list[count].y)%2==0) list[count].z=1;
				else list[count].z=-1;
			}
			count++;
		}

}

//MASK[x,y,sign*bfactor]
void MkPosList(MASK *list, int nsam, float bfactor)
{
	int hnsamb=nsam/2+1;
	int hnsam=nsam/2;
	int i,j;
	int count=0;
	int r2;
	float m=-0.5*bfactor/nsam/nsam;
	for(j=0;j<hnsam;j++)
		for(i=0;i<hnsamb;i++)
		{
			list[count].x=i;
			list[count].y=j;

			r2=list[count].x*list[count].x+list[count].y*list[count].y;
			if((list[count].x+list[count].y)%2==0) list[count].z=exp(m*r2);
			else list[count].z=-exp(m*r2);

			count++;
		}
	for(j=hnsam;j<nsam;j++)
		for(i=0;i<hnsamb;i++)
		{
			list[count].x=i;
			list[count].y=j-nsam;
			r2=list[count].x*list[count].x+list[count].y*list[count].y;
			if((list[count].x+list[count].y)%2==0) list[count].z=exp(m*r2);
			else list[count].z=-exp(m*r2);

			count++;
		}

}

__global__ void cuShiftCC(hipComplex *dfft, hipComplex *dsum, MASK *dposlist,float shx, float shy, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;

	MASK pos=dposlist[id];
	float shift=shx*pos.x+shy*pos.y;
	hipComplex phase;
	phase.x=cos(shift);
	phase.y=sin(shift);

	hipComplex val=cXc(dsum[id],conj(cXc(dfft[id],phase)));
	val.x/=size;
	val.x*=pos.z;
	val.y/=size;
	val.y*=pos.z;
	dfft[id]=val;

}
void GPUShiftCC(float *dfft, float *dsum, MASK *dposlist, float sx, float sy, int nsam)
{
	float shx=sx*2.0*PI/nsam;
	float shy=sy*2.0*PI/nsam;

	int size=(nsam/2+1)*nsam;
	cuShiftCC<<<size/BLOCKSIZE+1,BLOCKSIZE>>>((hipComplex *)dfft, (hipComplex *)dsum, dposlist,shx, shy, size);

}

__global__ void cuShift(hipComplex *dfft,MASK *dposlist,float shx, float shy, int size)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=size) return;

	MASK pos=dposlist[id];
	float shift=shx*pos.x+shy*pos.y;
	hipComplex phase;
	phase.x=cos(shift);
	phase.y=sin(shift);

	dfft[id]=cXc(dfft[id],phase);

}
void GPUShift(float *dfft, MASK *dposlist, float sx, float sy, int nsam)
{
	float shx=sx*2.0*PI/nsam;
	float shy=sy*2.0*PI/nsam;

	int size=(nsam/2+1)*nsam;
	cuShift<<<size/BLOCKSIZE+1,BLOCKSIZE>>>((hipComplex *)dfft,dposlist,shx, shy, size);

}

float FindShift(float *dsrc,int nsam, float* hboxmap, int box, float &sx, float &sy, int wNoise)
{
	int ori=(nsam-box)/2;
	int nsamb=nsam+2;
	int i,j,id,is;
	float bestcc=-1e9;

	//float *dst=new float[box*box];
	for(i=0;i<box;i++)
	{
		hipMemcpy(hboxmap+i*box,dsrc+(i+ori)*nsamb+ori,sizeof(float)*box,hipMemcpyDeviceToHost);
	}

	sx=0;
	sy=0;
	for(j=0;j<box;j++)
		for(i=0;i<box;i++)
		{
			id=j*box+i;

			if(abs(i-box/2)<=wNoise && abs(j-box/2)<=wNoise) continue;

			if(hboxmap[id]>bestcc)
			{
				bestcc=hboxmap[id];
				sx=i;
				sy=j;
			}
		}


	//Fourier interpolation
	int subbox=16; //box/4;
	float *hsubboxmap=new float[(subbox+2)*subbox];
	int offsetx=sx-subbox/2;
	int offsety=sy-subbox/2;
	if((offsetx+subbox)>box) offsetx=box-subbox;
	else if(offsetx<0) offsetx=0;
	if((offsety+subbox)>box) offsety=box-subbox;
	else if(offsety<0) offsety=0;
	//crop and fft
	crop2fft(hboxmap,box,hsubboxmap,offsetx,offsety,subbox);
	fft2d(hsubboxmap,subbox);
	cosmask2d((complex<float> *)hsubboxmap,subbox);
	//pad
	int scale=32;
	int wNoiseScaled=scale*wNoise;
	int pad=subbox*scale;
	float *hpadmap=new float[(pad+2)*pad];
	memset(hpadmap,0,sizeof(float)*(pad+2)*pad);
	for(i=0;i<subbox/2;i++)
	{
		memcpy(hpadmap+i*(pad+2),hsubboxmap+i*(subbox+2),sizeof(float)*(subbox+2));
	}
	for(i=0;i<subbox/2;i++)
	{
		memcpy(hpadmap+(pad-1-i)*(pad+2),hsubboxmap+(subbox-1-i)*(subbox+2),sizeof(float)*(subbox+2));
	}
	//ifft
	ifft2d(hpadmap,pad);
	//find shift
	int ox=(box/2-offsetx)*scale; //in order to avoid noise peak at box/2
	int oy=(box/2-offsety)*scale; //in order to avoid noise peak at box/2
	int sxp=0,syp=0;
	bestcc=hpadmap[0];
	for(j=0;j<pad;j++)
	{
		is=j*(pad+2);
		for(i=0;i<pad;i++)
		{
			if(abs(i-ox)<=wNoiseScaled && abs(j-oy)<=wNoiseScaled) continue;

			id=is+i;
			if(hpadmap[id]>bestcc)
			{
				bestcc=hpadmap[id];
				sxp=i;
				syp=j;
			}
		}
	}

	sx=offsetx+sxp/double(scale);
	sy=offsety+syp/double(scale);
	sx-=box/2;
	sy-=box/2;

	/*char filename[256];
	sprintf(filename,"D:\\UCSFImage\\DoseFragProcess\\data\\temp.mrc");
	MRC mrc;
	mrc.open(filename,"wb");
	mrc.createMRC(hpadmap,pad+2,pad,1);
	mrc.close();*/



	delete [] hsubboxmap;
	delete [] hpadmap;
	

	return bestcc/nsam/nsam/subbox/subbox;
}

float FindShift(float* hboxmap, int box, float &sx, float &sy)
{
	int i,j,id;
	float bestcc=-1e9;

	sx=0;
	sy=0;
	for(j=0;j<box;j++)
		for(i=0;i<box;i++)
		{
			id=j*box+i;
			if(hboxmap[id]>bestcc)
			{
				bestcc=hboxmap[id];
				sx=i;
				sy=j;
			}
		}
	sx-=box/2;
	sy-=box/2;

	return bestcc;
}

void testCUFFT()
{
	int i,j;
	int nsam=26;
	int size=(nsam+2)*nsam;
	float *h=new float[size];
	float *r=new float[size];
	memset(h,0,size*sizeof(float));
	memset(r,0,size*sizeof(float));
	for(j=0;j<nsam;j++)
		for(i=0;i<nsam;i++)
		{
			h[j*(nsam+2)+i]=i+1;
		}

	float *d=0;
	
	GPUMemAlloc((void **)&d,size*sizeof(float));
	GPUMemH2D(d,h,size*sizeof(float));

	hipfftHandle fft_plan=GPUFFTPlan(nsam);
	hipfftHandle ifft_plan=GPUIFFTPlan(nsam);

	GPUFFT2d(d,fft_plan);
	GPUSync();
	GPUIFFT2d(d,ifft_plan);
	GPUSync();

	GPUMultiplyNum(d,1.0/nsam/nsam,size);
	GPUMemD2H(r,d,sizeof(float)*size);

	char hstr[65536]="";
	char rstr[65536]="";
	char str[16]="";
	for(j=0;j<nsam;j++)
	{
		strcat(hstr,"\n");
		strcat(rstr,"\n");
		for(i=0;i<(nsam+2);i++)
		{
			sprintf(str,"%6.3f ",h[j*(nsam+2)+i]);
			strcat(hstr,str);
			sprintf(str,"%6.3f ",r[j*(nsam+2)+i]);
			strcat(rstr,str);
		}
		
	}
	
	GPUMemFree((void **)&d);
	GPUFFTDestroy(fft_plan);
	GPUFFTDestroy(ifft_plan);
	delete [] h;
	delete [] r;
	
/*GPUFFT2d(d,fft_plan);
	GPUSync();
	GPUIFFT2d(d,ifft_plan);
	GPUSync();*/
	return;
}
